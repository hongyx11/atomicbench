#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include "util.cuh"

int main(int argc, char *argv[]){
  printf ("\n This benchmark computes reduce operation using atomic kernel / customized kernel, \n"
  " on NVIDIA P100 / V100 / A100. we benchmark on scalar. \n\n");
  size_t n = atoi(argv[1]);
  printf(" Number of problem size %lu whole problem szie takes memory %.5f GB \n", n, n * 8./1000./1000./1000.);
  real * hiarr = new real[n];
  // init rnd data
  initrandomdata(hiarr, n);
  real cpusum = cpureduce(hiarr, n);
  printf(" cpu sum is %f \n", cpusum);
  // prepare gpu memory
  real * diarr, *doarr; // gpu ptr
  real * hout = new real[128]; // cpu ptr
  size_t redcueslot = 128;
  size_t paramsize = 1;
  cudanew(&diarr, n); cudanew(&doarr, redcueslot * paramsize);
  cudamemcpy(diarr, hiarr, n);
  // prepare gpu kernel size
  size_t bx = 128;
  size_t gx = n / bx + (n%bx != 0);
  
  printf("\n we are using atomic operation \n");

  hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start);
  hipDeviceSynchronize();
  atomickernel<<<gx,bx>>>(diarr,doarr);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  showstat(milliseconds * 1e-3, n);
  // get results back to cpu
  cudamemcpy(hout, doarr, 128);
  real gpusum = 0.0;
  for(int j=0; j<128;j++) gpusum += hout[j];
  printf(" gpu sum is %f , diff with cpu is %e \n", gpusum, abs((gpusum - cpusum))/n );
  cudadelete(diarr);
  cudadelete(doarr);
  delete[] hiarr;
  delete[] hout;
}